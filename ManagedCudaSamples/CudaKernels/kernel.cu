#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

// Texture reference
texture<float2, 2> texref;
 
extern "C"  {
	//kernel code
	__global__ void kernel(/* parameters */)
	{
				
	}
}